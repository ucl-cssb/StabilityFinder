#include "hip/hip_runtime.h"
#define NSPECIES 2
#define NPARAM 8
#define NREACT 2

#define gx tex2D(param_tex,0,tid)

#define kx tex2D(param_tex,1,tid)

#define nXY tex2D(param_tex,2,tid)

#define xXY tex2D(param_tex,3,tid)

#define lXY tex2D(param_tex,4,tid)

#define nXX tex2D(param_tex,5,tid)

#define xXX tex2D(param_tex,6,tid)

#define lXX tex2D(param_tex,7,tid)

__device__ double HS(double x, double xI, double nI, double lI ){
   return 1/(1 + pow(x/xI,nI)) + lI*(1 - 1/(1 + pow(x/xI,nI)) );
}


struct myFex{
    __device__ void operator()(int *neq, double *t, double *y, double *ydot/*, void *otherData*/){

        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        ydot[0] = gx*HS( y[1], xXY, nXY, lXY )*HS( y[0], xXX, nXX, lXX ) - kx*y[0]; //*HS( y[0], xXX, nXX, lXX );
        ydot[1] = gx*HS( y[0], xXY, nXY, lXY )*HS( y[1], xXX, nXX, lXX ) - kx*y[1]; //*HS( y[1], xXX, nXX, lXX );
    }
};

 struct myJex{
    __device__ void operator()(int *neq, double *t, double *y, int ml, int mu, double *pd, int nrowpd/*, void *otherData*/){
        return;
    }
};